#include "hip/hip_runtime.h"
#include <chrono>

#include <iostream>
#include <stdio.h>
#include <stdlib.h>


#include "gpu_convolution.h"
#include "hip/hip_runtime.h"

#define BLOCK_WIDTH 	32
#define BLOCK_HEIGHT	32

unsigned int divUp(const unsigned int& a, const unsigned int& b)
{
	if (a % b != 0) {
		return a / b + 1;
	}
	else {
		return a / b;
	}
}

const unsigned int MAX_FILTER_SIZE = 51;
__device__ __constant__ float d_cFilterKernel[MAX_FILTER_SIZE * MAX_FILTER_SIZE];

__global__ void applykernel(float* d_srcImagePtr, float* d_maskPtr, float* d_outImagePtr,
									int width, int height, int paddedWidth, int paddedHeight,
									int filterWidth, int filterHeight)
{
	const int s = floor(static_cast<float>(filterWidth) / 2);
	const int i = blockIdx.y * blockDim.y + threadIdx.y + s;
	const int j = blockIdx.x * blockDim.x + threadIdx.x + s;

	unsigned int filterRowIndex = 0;
	unsigned int srcImgRowIndex = 0;
	unsigned int srcImgIndex = 0;
	unsigned int maskIndex = 0;
	float pixelSum = 0;

	// Check out of bounds thread idx
	if( j >= s && j < paddedWidth - s &&
			i >= s && i < paddedHeight - s) {

		int outPixelPos = (j - s) + (i - s) * width;

		// Apply convolution
		for (int h = -s;  h <= s; h++) {
			filterRowIndex = (h + s) * filterWidth;
	    	srcImgRowIndex = (h + i) * paddedWidth;
	    	for (int w = -s; w <= s; w++) {
	    		srcImgIndex = w + j + srcImgRowIndex;
	    		maskIndex = (w + s) + filterRowIndex;
	    		pixelSum += d_srcImagePtr[srcImgIndex] * d_maskPtr[maskIndex];
	    	}
		}

		// Thresholding overflowing pixel's values
		if (pixelSum < 0) {
			pixelSum = 0;
		}
		else if (pixelSum > 255) {
			pixelSum = 255;
		}

		// Write pixel on the output image
		d_outImagePtr[outPixelPos] = pixelSum;
		pixelSum = 0;
	}
}

bool runKernel(const float* srcImage, float* outImage, const float* mask, int width, int height, 
                int paddedWidth, int paddedHeight, int filterWidth, int filterHeight) // magic happen here.
{
	std::cout << "Starting CUDA global memory convolution" << std::endl;

	const int blockWidth = BLOCK_WIDTH;
	const int blockHeight = BLOCK_HEIGHT;

	float *d_srcImagePtr;
	float *d_outImagePtr;
	float *d_maskPtr;

	const int srcImgSize = sizeof(float) * paddedWidth * paddedHeight;
	const int maskSize = sizeof(float) * filterWidth * filterHeight;
	const int outImageSize = sizeof(float) * width * height;

	int copyDuration = 0;
	auto t3 = std::chrono::high_resolution_clock::now();

	// Allocate device memory for images and filter
	hipMalloc(reinterpret_cast<void**>(&d_srcImagePtr), srcImgSize);
	hipMalloc(reinterpret_cast<void**>(&d_maskPtr), maskSize);
	hipMalloc(reinterpret_cast<void**>(&d_outImagePtr), outImageSize);

	auto t4 = std::chrono::high_resolution_clock::now();
	copyDuration += std::chrono::duration_cast<std::chrono::microseconds>(t4 - t3).count();

	// hipError_t err = hipGetLastError();

	t3 = std::chrono::high_resolution_clock::now();

	// Transfer data from host to device memory
	hipMemcpy(d_srcImagePtr, srcImage, srcImgSize, hipMemcpyHostToDevice);
	hipMemcpy(d_maskPtr, mask, maskSize, hipMemcpyHostToDevice);

	t4 = std::chrono::high_resolution_clock::now();
	copyDuration += std::chrono::duration_cast<std::chrono::microseconds>(t4 - t3).count();

	// Allocates block size and grid size
	dim3 threadsPerBlock(blockWidth, blockHeight);
	dim3 blocksPerGrid(divUp(width, blockWidth), divUp(height, blockHeight));

	auto t1 = std::chrono::high_resolution_clock::now();

	applykernel<<<blocksPerGrid, threadsPerBlock>>>(d_srcImagePtr, d_maskPtr, d_outImagePtr, width,  height,  paddedWidth,  paddedHeight, filterWidth,  filterHeight);

	// Waits for threads to finish work
	hipDeviceSynchronize();

	auto t2 = std::chrono::high_resolution_clock::now();
	auto filterDuration = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
	std::cout << "execution time: " << filterDuration << " μs" << std::endl;

	t3 = std::chrono::high_resolution_clock::now();

	// Transfer resulting image back
	hipMemcpy(outImage, d_outImagePtr, outImageSize, hipMemcpyDeviceToHost);

	t4 = std::chrono::high_resolution_clock::now();
	copyDuration += std::chrono::duration_cast<std::chrono::microseconds>(t4 - t3).count();
	std::cout << "Copy time: " << copyDuration << " μs" << std::endl;

	hipFree(d_srcImagePtr);
	hipFree(d_maskPtr);
	hipFree(d_outImagePtr);

	return true;
}