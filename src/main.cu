#include <iostream>
#include <chrono>
#include "image.h"
#include "kernel.h"

#define OUTPUT_FOLDER   "output/"
#define OUTPUT_EXT       ".png"

int main(int argc, char **argv)
{
	std::cout << "===== Multithread kernel convolution =====" << std::endl;

	// Check command line parameters
	if (argc < 4) {
		std::cerr << "Usage: " << argv[0] << " filter_type image_path cuda_mem_tye" << std::endl;
	    std::cerr << "image path " << std::endl;
	    std::cerr << "kernel size e.g. 3,5,7,..." << std::endl;
		std::cerr << "std for the gaussian kernel" << std::endl;
	    return 1;
	}

    Kernel filter = Kernel();
	filter.setGaussianFilter(atoi(argv[2]), atoi(argv[2]), std::stof(argv[3]));
	filter.printKernel();

	Image img;
	bool loadResult = img.loadImage(argv[1]);
	if (!loadResult) {
		std::cerr << "Unable to load image " << argv[1] << std::endl;
		return 1;
	}

	Image newMtImg;
	Image newNpImg;

	// Init the CUDA device
	hipFree(0);

	// Executing multithread filtering for each image
	auto t1 = std::chrono::high_resolution_clock::now();
	bool cudaResult = img.multithreadFilter(newMtImg, filter);
	auto t2 = std::chrono::high_resolution_clock::now();

	std::cout << std::endl;

	auto t3 = std::chrono::high_resolution_clock::now();
	bool Result = img.applyFilter(newNpImg, filter);
	auto t4 = std::chrono::high_resolution_clock::now();

	std::cout << std::endl;

	// Evaluating execution times and save results
	if (cudaResult) {
		auto multithreadDuration = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
		std::cout << "Total CUDA Execution time: " << multithreadDuration << " μs" << std::endl;
		newMtImg.saveImage(std::string(std::string(OUTPUT_FOLDER) +  "result" + std::string(OUTPUT_EXT)).c_str());
	}

	if (Result) {
		auto singleDuration = std::chrono::duration_cast<std::chrono::microseconds>(t4 - t3).count();
		std::cout << "Total CPU Execution time: " << singleDuration << " μs" << std::endl;
	}
}
